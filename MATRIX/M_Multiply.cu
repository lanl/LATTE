#include "hip/hip_runtime.h"
/*!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
! Copyright 2010.  Los Alamos National Security, LLC. This material was    !
! produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos !
! National Laboratory (LANL), which is operated by Los Alamos National     !
! Security, LLC for the U.S. Department of Energy. The U.S. Government has !
! rights to use, reproduce, and distribute this software.  NEITHER THE     !
! GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY,     !
! EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS         !
! SOFTWARE.  If software is modified to produce derivative works, such     !
! modified software should be clearly marked, so as not to confuse it      !
! with the version available from LANL.                                    !
!                                                                          !
! Additionally, this program is free software; you can redistribute it     !
! and/or modify it under the terms of the GNU General Public License as    !
! published by the Free Software Foundation; version 2.0 of the License.   !
! Accordingly, this program is distributed in the hope that it will be     !
! useful, but WITHOUT ANY WARRANTY; without even the implied warranty of   !
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General !
! Public License for more details.                                         !
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!*/

#include "Matrix.h"

extern hipblasHandle_t* handle;
extern int ndevices;
extern int nblocks;
extern hipStream_t stream[];
extern hipEvent_t event[];

// Matrix multiplication on multiple GPUs
void M_MultiplyMgpu(REAL *scalar1, Matrix A, Matrix B, REAL *scalar2, Matrix C) {

  int idevice = 0;          // GPU 0  
  int cdev;
  int ks;
  int iblock1, iblock2;     // indices for blocks to multiply    
  int oblock;               // index for output block
  int kblock, kpblock;      // which sub-block
  int sub = A.DN / nblocks; // size of each block

  // Save current device
  hipGetDevice(&cdev);

  //printf("DN = %d  nblocks = %d  sub = %d\n", A.DN, nblocks, sub);
  kblock = 0;
  ks = 0;

  for (int k = 0; k < nblocks; k++) {
    kpblock = 0;

    for (int kp = 0; kp < nblocks; kp++) {

      iblock1 = kblock * A.DN;
      iblock2 = kpblock * A.DN;

//      oblock = kp * sub * sub + k * A.DN * sub;

	oblock = kp * sub + k * A.DN * sub;

      idevice = k % ndevices;
      hipSetDevice(idevice);


//      printf("idevice = %d  iblock1 = %d  iblock2 = %d  oblock = %d\n", idevice, iblock1, iblock2, oblock);

       hipblasSetStream(handle[idevice], stream[idevice]);

      // Multiply - results in sub x sub block
#if REALSIZE==4

hipblasSgemm(handle[idevice], HIPBLAS_OP_T, HIPBLAS_OP_N, sub, sub, A.DN, scalar1, B.Device[idevice]+iblock2, B.DN, A.Devic\
e[idevice]+iblock1, A.DN, scalar2, C.Device[idevice]+oblock, C.DN);      

#elif REALSIZE==8

   hipblasDgemm(handle[idevice], HIPBLAS_OP_T, HIPBLAS_OP_N, sub, sub, A.DN, scalar1, B.Device[idevice]+iblock2, B.DN, A.Device[idevice]+iblock1, A.DN, scalar2, C.Device[idevice]+oblock, C.DN);

#endif

      ks++;
      kpblock += sub;
    }
    
    kblock += sub;
  }

/*
  // Assemble piece on each device
  for (int d = 0; d < ndevices; ++d) {
    hipSetDevice(d);
    M_AssembleMgpu(C, C2, sub, d);
  }

 // Add x0 back in for each device
  for (int d = 0; d < ndevices; ++d) {
    hipSetDevice(d);
    M_MultiplyScalarSumMgpu(scalar2, A, C, d);
  } */


  // Wait for devices and streams to finish
  M_Wait();

  // Restore device
  hipSetDevice(cdev);
}


void M_Multiply(Matrix A, Matrix B, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#endif

}

void M_Multiply3(Matrix A, Matrix B, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ONE, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ONE, C.Device[0], C.DM);
#endif

}

void M_MultiplyTranspose(Matrix A, Matrix B, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_T, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_T, HIPBLAS_OP_N, A.DM, B.DN, A.DN, &ONE, A.Device[0], A.DM, B.Device[0], B.DM, &ZERO, C.Device[0], C.DM);
#endif

}

void M_Multiply(REAL *scalar1, Matrix A, Matrix B, REAL *scalar2, Matrix C) {

  hipSetDevice(0);

#if REALSIZE==4
  hipblasSgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, scalar1, A.Device[0], A.DM, B.Device[0], B.DM, scalar2, C.Device[0], C.DM);
#elif REALSIZE==8
  hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, scalar1, A.Device[0], A.DM, B.Device[0], B.DM, scalar2, C.Device[0], C.DM);
#endif

}

void M_Multiply(int tposea, int tposeb, REAL *alpha, Matrix A, Matrix B, REAL *beta, Matrix C) {

     hipSetDevice(0);

//     printf("tposea = %d  tposeb  = %d \n", tposea, tposeb);
     if (tposea == 0 && tposeb == 0 ) {
       hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_N, A.DM, B.DN, A.DN, alpha, 
		   A.Device[0], A.DM, B.Device[0], B.DM, beta, C.Device[0], C.DM);
     } else if (tposea == 1 && tposeb == 0 ) {
       hipblasDgemm(handle[0], HIPBLAS_OP_T, HIPBLAS_OP_N, A.DM, B.DN, A.DN, alpha,
                   A.Device[0], A.DM, B.Device[0], B.DM, beta, C.Device[0], C.DM);
     } else if (tposea == 0 && tposeb == 1 ) {
       hipblasDgemm(handle[0], HIPBLAS_OP_N, HIPBLAS_OP_T, A.DM, B.DN, A.DN, alpha,
                   A.Device[0], A.DM, B.Device[0], B.DM, beta, C.Device[0], C.DM);
     }

}     

void M_Multiply(REAL k, Matrix A, Matrix B) {

  int msize = A.DM * A.DN;
  int size = msize >> 1;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  MultiplyScalarMatrixKernel<<<blockCount,NUM_THREADS>>>(msize, k, A.Device[0], B.Device[0]);

}

void M_MultiplyAdd(REAL k, Matrix A, REAL k2,  Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  MultiplyScalarMatrixAddKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], k2, B.Device[0], C.Device[0]);
}

void M_MultiplyAdd(REAL k, Matrix A, Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  hipSetDevice(0);

  MultiplyScalarMatrixAddMatrixKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], B.Device[0], C.Device[0]);
}

void M_MultiplySub(REAL k, Matrix A, REAL k2,  Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  hipSetDevice(0);

  MultiplyScalarMatrixSubKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], k2, B.Device[0], C.Device[0]);
}

void M_MultiplySub(REAL k, Matrix A, Matrix B, Matrix C) {

  int size = A.DM * A.DN;
  int blockCount = (int) ceil((float)size/(float)NUM_THREADS);

  hipSetDevice(0);

  MultiplyScalarMatrixSubMatrixKernel<<<blockCount,NUM_THREADS>>>(size, k, A.Device[0], B.Device[0], C.Device[0]);
}

